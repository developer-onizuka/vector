#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <fcntl.h>
#include <unistd.h>

#include "cufile.h"

#define KB(x) ((x)*1024L)
#define N 8

__global__ void vector_sqrt(float *s, float *t, float *u) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        u[idx] = sqrt(s[idx]*s[idx] + t[idx]*t[idx]);
        /* printf("idx:%d, blockId.x:%d, threadIdx.x:%d\n",idx, blockIdx.x, threadIdx.x); */
}

int main(int argc, char *argv[])
{
	FILE *fpa,*fpb;
	int fpc, fpd;
	float *a,*b,*c,*d,*x;
	float *c_d,*d_d,*x_d;
	int n;
        CUfileDescr_t cf_desc_c;
        CUfileDescr_t cf_desc_d;
        CUfileHandle_t cf_handle_c;
        CUfileHandle_t cf_handle_d;
	if(argc < 2) {
		n = N;
	} else {
		n = atoi(argv[1]);
	}
	a = (float*)malloc(sizeof(float)*n);
	b = (float*)malloc(sizeof(float)*n);
	for(int i=0;i<n;++i) {
		a[i] = 3.0;
		b[i] = 4.0;
	}

	fpa = fopen("./float_a.bin", "wr");
	fpb = fopen("./float_b.bin", "wr");
	fwrite(a, sizeof(float), n, fpa);
	fwrite(b, sizeof(float), n, fpb);
	fclose(fpa);
	fclose(fpb);

	c = (float*)malloc(sizeof(float)*n);
	d = (float*)malloc(sizeof(float)*n);
	x = (float*)malloc(sizeof(float)*n);
	hipMalloc(&c_d, sizeof(float)*n);
	hipMalloc(&d_d, sizeof(float)*n);
	hipMalloc(&x_d, sizeof(float)*n);

        cuFileDriverOpen();
        fpc = open("./float_a.bin", O_RDONLY | O_DIRECT);
        fpd = open("./float_b.bin", O_RDONLY | O_DIRECT);
        cf_desc_c.handle.fd = fpc;
        cf_desc_d.handle.fd = fpd;
        cf_desc_c.type = CU_FILE_HANDLE_TYPE_OPAQUE_FD;
        cf_desc_d.type = CU_FILE_HANDLE_TYPE_OPAQUE_FD;

        cuFileHandleRegister(&cf_handle_c, &cf_desc_c);
        cuFileHandleRegister(&cf_handle_d, &cf_desc_d);
        cuFileBufRegister((float*)c_d, sizeof(float)*n, 0);
        cuFileBufRegister((float*)d_d, sizeof(float)*n, 0);

	cuFileRead(cf_handle_c, (float*)c_d, sizeof(float)*n, 0, 0);
	cuFileRead(cf_handle_d, (float*)d_d, sizeof(float)*n, 0, 0);
	hipMemcpy(x_d, x, sizeof(float)*n, hipMemcpyHostToDevice);

        int blocksize = 512;
        int gridsize = (n+(blocksize-1))/blocksize;
        dim3 dimGrid(gridsize,1);
        dim3 dimBlock(blocksize,1,1);
        vector_sqrt<<<dimGrid,dimBlock>>>(c_d,d_d,x_d);

	hipMemcpy(x, x_d, sizeof(float)*n, hipMemcpyDeviceToHost);
	for(int i=0;i<n;++i) {
		printf("output: %5.3e\n", x[i]);
	}
	/* printf("dimGrid:%d, dimBlock:%d\n", gridsize, blocksize); */

	cuFileBufDeregister((float*)c_d);
	cuFileBufDeregister((float*)d_d);

	free(a);
	free(b);
	free(c);
	free(d);
	free(x);
	hipFree(c_d);
	hipFree(d_d);
	hipFree(x_d);

	close(fpc);
	close(fpd);
	cuFileDriverClose();
}
