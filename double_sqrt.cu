#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>


#define KB(x) ((x)*1024L)
#define N 8

__global__ void vector_sqrt(double *s, double *t, double *u, int n) {
	for(int i=0;i<n;i++) {
		u[i] = sqrt(s[i]*s[i] + t[i]*t[i]);
	}
}

int main(int argc, char *argv[])
{
	FILE *fpa,*fpb,*fpc,*fpd;
	double *a,*b,*c,*d,*x;
	double *c_d,*d_d,*x_d;
	int n;
	if(argc < 2) {
		n = N;
	} else {
		n = atoi(argv[1]);
	}
	a = (double*)malloc(sizeof(double)*n);
	b = (double*)malloc(sizeof(double)*n);
	for(int i=0;i<n;++i) {
		a[i] = 3.0;
		b[i] = 4.0;
	}

	fpa = fopen("./double_a.bin", "wr");
	fpb = fopen("./double_b.bin", "wr");
	fwrite(a, sizeof(double), n, fpa);
	fwrite(b, sizeof(double), n, fpb);
	fclose(fpa);
	fclose(fpb);

	c = (double*)malloc(sizeof(double)*n);
	d = (double*)malloc(sizeof(double)*n);
	x = (double*)malloc(sizeof(double)*n);
	hipMalloc(&c_d, sizeof(double)*n);
	hipMalloc(&d_d, sizeof(double)*n);
	hipMalloc(&x_d, sizeof(double)*n);
	fpc = fopen("./double_a.bin", "r");
	fpd = fopen("./double_b.bin", "r");
	fread(c, sizeof(double), n, fpc);
	fread(d, sizeof(double), n, fpd);
	hipMemcpy(c_d, c, sizeof(double)*n, hipMemcpyHostToDevice);
	hipMemcpy(d_d, d, sizeof(double)*n, hipMemcpyHostToDevice);
	hipMemcpy(x_d, x, sizeof(double)*n, hipMemcpyHostToDevice);
	vector_sqrt<<<2,128>>>(c_d,d_d,x_d,n);
	hipMemcpy(x, x_d, sizeof(double)*n, hipMemcpyDeviceToHost);
	for(int i=0;i<n;++i) {
		printf("output: %8.3lf\n", x[i]);
	}

	free(a);
	free(b);
	free(c);
	free(d);
	free(x);
	hipFree(c_d);
	hipFree(d_d);
	hipFree(x_d);

	fclose(fpc);
	fclose(fpd);
}
