#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <fcntl.h>
#include <unistd.h>

#include "cufile.h"

#define KB(x) ((x)*1024L)
#define N 8

__global__ void vector_sqrt(double *s, double *t, double *u) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        u[idx] = sqrt(s[idx]*s[idx] + t[idx]*t[idx]);
        /* printf("idx:%d, blockId.x:%d, threadIdx.x:%d\n",idx, blockIdx.x, threadIdx.x); */
}

int main(int argc, char *argv[])
{
	FILE *fpa,*fpb;
	int fpc, fpd;
	double *a,*b,*c,*d,*x;
	double *c_d,*d_d,*x_d;
	int n;
        CUfileDescr_t cf_desc_c;
        CUfileDescr_t cf_desc_d;
        CUfileHandle_t cf_handle_c;
        CUfileHandle_t cf_handle_d;
	if(argc < 2) {
		n = N;
	} else {
		n = atoi(argv[1]);
	}
	a = (double*)malloc(sizeof(double)*n);
	b = (double*)malloc(sizeof(double)*n);
	for(int i=0;i<n;++i) {
		a[i] = 3.0;
		b[i] = 4.0;
	}

	fpa = fopen("./double_a.bin", "wr");
	fpb = fopen("./double_b.bin", "wr");
	fwrite(a, sizeof(double), n, fpa);
	fwrite(b, sizeof(double), n, fpb);
	fclose(fpa);
	fclose(fpb);

	c = (double*)malloc(sizeof(double)*n);
	d = (double*)malloc(sizeof(double)*n);
	x = (double*)malloc(sizeof(double)*n);
	hipMalloc(&c_d, sizeof(double)*n);
	hipMalloc(&d_d, sizeof(double)*n);
	hipMalloc(&x_d, sizeof(double)*n);

        cuFileDriverOpen();
        fpc = open("./double_a.bin", O_RDONLY | O_DIRECT);
        fpd = open("./double_b.bin", O_RDONLY | O_DIRECT);
        cf_desc_c.handle.fd = fpc;
        cf_desc_d.handle.fd = fpd;
        cf_desc_c.type = CU_FILE_HANDLE_TYPE_OPAQUE_FD;
        cf_desc_d.type = CU_FILE_HANDLE_TYPE_OPAQUE_FD;

        cuFileHandleRegister(&cf_handle_c, &cf_desc_c);
        cuFileHandleRegister(&cf_handle_d, &cf_desc_d);
        cuFileBufRegister((double*)c_d, sizeof(double)*n, 0);
        cuFileBufRegister((double*)d_d, sizeof(double)*n, 0);

	cuFileRead(cf_handle_c, (double*)c_d, sizeof(double)*n, 0, 0);
	cuFileRead(cf_handle_d, (double*)d_d, sizeof(double)*n, 0, 0);
	hipMemcpy(x_d, x, sizeof(double)*n, hipMemcpyHostToDevice);

        int blocksize = 512;
        int gridsize = (n+(blocksize-1))/blocksize;
        dim3 dimGrid(gridsize,1);
        dim3 dimBlock(blocksize,1,1);
        vector_sqrt<<<dimGrid,dimBlock>>>(c_d,d_d,x_d);

	hipMemcpy(x, x_d, sizeof(double)*n, hipMemcpyDeviceToHost);
	for(int i=0;i<n;++i) {
		printf("output: %8.3lf\n", x[i]);
	}
	/* printf("dimGrid:%d, dimBlock:%d\n", gridsize, blocksize); */

	cuFileBufDeregister((double*)c_d);
	cuFileBufDeregister((double*)d_d);

	free(a);
	free(b);
	free(c);
	free(d);
	free(x);
	hipFree(c_d);
	hipFree(d_d);
	hipFree(x_d);

	close(fpc);
	close(fpd);
	cuFileDriverClose();
}
