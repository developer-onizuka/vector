#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <fcntl.h>
#include <unistd.h>

#include "cufile.h"

#define KB(x) ((x)*1024L)
#define N 8

__global__ void vector_sqrt(double *s, double *t, double *u) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        u[idx] = sqrt(s[idx]*s[idx] + t[idx]*t[idx]);
        /* printf("idx:%d, blockId.x:%d, threadIdx.x:%d\n",idx, blockIdx.x, threadIdx.x); */
}

int main(int argc, char *argv[])
{
	FILE *fpa,*fpb,*fpz;
	int fpc, fpd, fpx;
	double *a,*b,*z,*c,*d,*x;
	double *c_d,*d_d,*x_d;
	int n;
        CUfileDescr_t cf_desc_c;
        CUfileDescr_t cf_desc_d;
        CUfileDescr_t cf_desc_x;
        CUfileHandle_t cf_handle_c;
        CUfileHandle_t cf_handle_d;
        CUfileHandle_t cf_handle_x;
	if(argc < 2) {
		n = N;
	} else {
		n = atoi(argv[1]);
	}
	a = (double*)malloc(sizeof(double)*n);
	b = (double*)malloc(sizeof(double)*n);
	z = (double*)malloc(sizeof(double)*n);
	for(int i=0;i<n;++i) {
		a[i] = 3.0;
		b[i] = 4.0;
		z[i] = 0.0;
	}

	fpa = fopen("./double_a.bin", "wr");
	fpb = fopen("./double_b.bin", "wr");
	fpz = fopen("./double_z.bin", "wr");
	fwrite(a, sizeof(double), n, fpa);
	fwrite(b, sizeof(double), n, fpb);
	fwrite(z, sizeof(double), n, fpz);
	fclose(fpa);
	fclose(fpb);
	fclose(fpz);

	c = (double*)malloc(sizeof(double)*n);
	d = (double*)malloc(sizeof(double)*n);
	x = (double*)malloc(sizeof(double)*n);
	hipMalloc(&c_d, sizeof(double)*n);
	hipMalloc(&d_d, sizeof(double)*n);
	hipMalloc(&x_d, sizeof(double)*n);

        cuFileDriverOpen();
        fpc = open("./double_a.bin", O_RDONLY | O_DIRECT);
        fpd = open("./double_b.bin", O_RDONLY | O_DIRECT);
        fpx = open("./double_z.bin", O_RDWR | O_DIRECT);
        cf_desc_c.handle.fd = fpc;
        cf_desc_d.handle.fd = fpd;
        cf_desc_x.handle.fd = fpx;
        cf_desc_c.type = CU_FILE_HANDLE_TYPE_OPAQUE_FD;
        cf_desc_d.type = CU_FILE_HANDLE_TYPE_OPAQUE_FD;
        cf_desc_x.type = CU_FILE_HANDLE_TYPE_OPAQUE_FD;

        cuFileHandleRegister(&cf_handle_c, &cf_desc_c);
        cuFileHandleRegister(&cf_handle_d, &cf_desc_d);
        cuFileHandleRegister(&cf_handle_x, &cf_desc_x);
        cuFileBufRegister((double*)c_d, sizeof(double)*n, 0);
        cuFileBufRegister((double*)d_d, sizeof(double)*n, 0);
        cuFileBufRegister((double*)x_d, sizeof(double)*n, 0);

	cuFileRead(cf_handle_c, (double*)c_d, sizeof(double)*n, 0, 0);
	cuFileRead(cf_handle_d, (double*)d_d, sizeof(double)*n, 0, 0);
	cuFileRead(cf_handle_x, (double*)x_d, sizeof(double)*n, 0, 0);
	/* cudaMemcpy(x_d, x, sizeof(double)*n, cudaMemcpyHostToDevice); */

        int blocksize = 512;
        int gridsize = (n+(blocksize-1))/blocksize;
        dim3 dimGrid(gridsize,1);
        dim3 dimBlock(blocksize,1,1);
        vector_sqrt<<<dimGrid,dimBlock>>>(c_d,d_d,x_d);

	cuFileWrite(cf_handle_x, (double*)x_d, sizeof(double)*n, 0, 0);
	/* cudaMemcpy(x, x_d, sizeof(double)*n, cudaMemcpyDeviceToHost);
	for(int i=0;i<n;++i) {
		printf("output: %8.3lf\n", x[i]);
	} */
	/* printf("dimGrid:%d, dimBlock:%d\n", gridsize, blocksize); */

	cuFileBufDeregister((double*)c_d);
	cuFileBufDeregister((double*)d_d);
	cuFileBufDeregister((double*)x_d);

	free(a);
	free(b);
	free(c);
	free(d);
	free(x);
	hipFree(c_d);
	hipFree(d_d);
	hipFree(x_d);

	close(fpc);
	close(fpd);
	close(fpx);
	cuFileDriverClose();
}
